#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_THREAD 1024

template <typename scalar_t>
__global__ void _forward_kernel(scalar_t* x, scalar_t* y, const int size) {
    const uint32_t index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < size) {
        y[index] = 1 / (1 + exp(-x[index]));
    }
}

template <typename scalar_t>
__global__ void _backward_kernel(scalar_t* y, scalar_t* dy, scalar_t* g, const int size) {
    const uint32_t index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < size) {
        g[index] = dy[index] * y[index] * (1 - y[index]);
    }
}

__host__ at::Tensor sigmoid_forward_cuda(at::Tensor& x) {
    const int size = x.numel();
    const int nthred = NUM_THREAD;
    const int nblock = (size + nthred - 1) / nthred;

    auto y = at::empty_like(x); // Create an output tensor y with the same shape as x

    // Dispatch the appropriate scalar type at runtime
    AT_DISPATCH_FLOATING_TYPES(
        x.scalar_type(), "sigmoid_forward_cuda", 
        [&]() {
            _forward_kernel<scalar_t><<<nblock, nthred>>>(
                x.data<scalar_t>(), y.data<scalar_t>(), size
    );});

    return y;
}

__host__ at::Tensor sigmoid_backward_cuda(const at::Tensor& y, const at::Tensor& dy) {
    const int size = y.numel();
    const int nthred = NUM_THREAD;
    const int nblock = (size + nthred - 1) / nthred;

    auto g = dy.clone(); // Clone dy to create the output tensor g

    // Dispatch the appropriate scalar type at runtime
    AT_DISPATCH_FLOATING_TYPES(
        dy.scalar_type(), "sigmoid_backward_cuda",
        [&]() {
        _backward_kernel<scalar_t><<<nblock, nthred>>>(
            y.data<scalar_t>(), dy.data<scalar_t>(),
            g.data<scalar_t>(), size
    );});

    return g;
}