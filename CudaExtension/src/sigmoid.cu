#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024

template <typename scalar_t>
__global__ void sigmoid_forward_kernel(const scalar_t* __restrict__ input, scalar_t* __restrict__ output, const int n) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        auto expx = expf(-input[index]);
        output[index] = expx / (1.0 + expx);
    }
}

template <typename scalar_t>
__global__ void sigmoid_backward_kernel(const scalar_t* __restrict__ grad_output, const scalar_t* __restrict__ output, scalar_t* __restrict__ grad_input, const int n) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        grad_input[index] = grad_output[index] * output[index] * (1.0 - output[index]);
    }
}

__host__ at::Tensor sigmoid_forward_cuda(const at::Tensor& input) {
    const auto n = input.numel();
    auto output = at::empty_like(input);
    const dim3 blocks((n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    const dim3 threads(THREADS_PER_BLOCK);
    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "sigmoid_forward_cuda", ([&] {
        sigmoid_forward_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            n
        );
    }));
    return output;
}

__host__ at::Tensor sigmoid_backward_cuda(const at::Tensor& grad_output, const at::Tensor& output) {
    const auto n = grad_output.numel();
    auto grad_input = at::empty_like(grad_output);
    const dim3 blocks((n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    const dim3 threads(THREADS_PER_BLOCK);
    AT_DISPATCH_FLOATING_TYPES(grad_output.scalar_type(), "sigmoid_backward_cuda", ([&] {
        sigmoid_backward_kernel<scalar_t><<<blocks, threads>>>(
            grad_output.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            grad_input.data_ptr<scalar_t>(),
            n
        );
    }));
    return grad_input;
}